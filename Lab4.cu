
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <assert.h>
#include <unistd.h>



using namespace std;

inline hipError_t HANDLE_ERROR(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess)
  {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
    return result;
}



__global__ void computeAverageGPU(double *hArray, double *gArray, int iteration, int width)
{

    // Calculate the column index of the Pd element, denote by x
    int m = threadIdx.x + blockIdx.x * blockDim.x;
    // Calculate the row index of the Pd element, denote by y
    int n = threadIdx.y + blockIdx.y * blockDim.y;

    if (m > 0 && m < (width-1) && n > 0 && n < (width-1))
    {
        gArray[m*width + n] = 0.25*(hArray[m*width + n - 1] + hArray[m*width + n + 1] + hArray[(m*width + n) + width] + hArray[(m*width + n) - width]);
    }

}



int main(int argc, char* argv[]) {

    //TODO: Check user input is in correct format.
    int nFlags, iFlags, opt;
    int iNumber, nNumber;

    nFlags = 0;
    iFlags = 0;

    string tempOptArg;
    while ((opt = getopt(argc, argv, "n:I:")) != -1)
    {
        switch (opt)
        {
            case 'n':
                // Do something
                tempOptArg = optarg;
                for (int i = 0; i < tempOptArg.length(); ++i)
                {
                    if (!isdigit(tempOptArg[i]))
                    {
                        printf("Invalid Input!\n");
                        return 0;
                    }
                }
                if (tempOptArg[0] == '0' && tempOptArg.length() > 1)
                {
                    printf("Invalid Input!\n");
                    return 0;
                }
                nFlags = 1;
                nNumber = atoi(optarg);
                break;
            case 'I':
                // Do something
                tempOptArg = optarg;
                for (int i = 0; i < tempOptArg.length(); ++i)
                {
                    if (!isdigit(tempOptArg[i]))
                    {
                        printf("Invalid Input!\n");
                        return 0;
                    }
                }

                if (tempOptArg[0] == '0' && tempOptArg.length() > 1)
                {
                    printf("Invalid Input!\n");
                    return 0;
                }
                iFlags = 1;
                iNumber = atoi(optarg);
                break;

            default: /* '?' */
                printf("Invalid Input!\n");
                return 0;
        }
    }

    if(argc != 5)
    {
        printf("Invalid Input!\n");
        return 0;
    }
    if (iNumber <= 0 || nNumber <= 0)
    {
        printf("Invalid Input!\n");
        return 0;
    }

    if (iFlags == 0 || nFlags == 0)
    {
        printf("Invalid Input!\n");
        return 0;
    }


    ofstream finalTemperatures;
    finalTemperatures.open("finalTemperatures.csv");




    int width = nNumber;
    int numIterations = iNumber / 2;

    int exteriorWidth = width + 2;

    int size = (width + 2) * (width + 2) * sizeof(double);

    double *gArray, *hArray;


    // capture start time
    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );

    hipMallocManaged(&hArray, size*sizeof(double));
    hipMallocManaged(&gArray, size*sizeof(double));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    int blockSize = sqrt(prop.maxThreadsPerBlock);
//    int numBlocks = (exteriorWidth + 32 - 1) / 32;
    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid((width/blockSize) + 1, (width/blockSize) + 1);

//

    int hotPlateStart = (exteriorWidth - (exteriorWidth * .4)) / 2;
    int hotPlateEnd = hotPlateStart + (exteriorWidth * .4);

    for(int i = 0; i < size; ++i)
    {
        hArray[i] = 0;
        gArray[i] = 0;
    }


    for (int i = 0; i < exteriorWidth; ++i)
    {
        hArray[i * exteriorWidth] = 20;
        hArray[(i * exteriorWidth + exteriorWidth) - 1] = 20;
        hArray[(exteriorWidth * exteriorWidth) - exteriorWidth + i] = 20;
        hArray[i] = 20;

        gArray[i * exteriorWidth] = 20;
        gArray[(i * exteriorWidth + exteriorWidth) - 1] = 20;
        gArray[(exteriorWidth * exteriorWidth) - exteriorWidth + i] = 20;
        gArray[i] = 20;



        if (i >= hotPlateStart && i < hotPlateEnd)
        {
            gArray[i] = 100;
            hArray[i] = 100;
        }

    }

    for (int iteration = 0; iteration < numIterations; ++iteration)
    {
        computeAverageGPU<<<dimGrid, dimBlock>>>(hArray, gArray, iteration, exteriorWidth);
        hipDeviceSynchronize();
        computeAverageGPU<<<dimGrid, dimBlock>>>(gArray, hArray, iteration, exteriorWidth);
    }


    hipDeviceSynchronize();

    // get stop time, and display the timing results
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
    printf( "Thin plate calculation took %3.3f milliseconds.\n", elapsedTime );



    for(int m = 0; m < exteriorWidth; ++m)
    {
        for(int n = 0; n < exteriorWidth; ++n)
        {
            if (n == exteriorWidth - 1)
            {
                finalTemperatures << setprecision(15) << hArray[m*exteriorWidth + n];
                continue;
            }
            finalTemperatures << setprecision(15) << hArray[m*exteriorWidth + n] << ",";
        }
        finalTemperatures << "\n";
    }

    finalTemperatures.close();

    hipFree(gArray);
    hipFree(hArray);

    // destroy events to free memory
    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );

    return 0;
}
